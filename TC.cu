#include "hip/hip_runtime.h"
#include <iostream>
#include "graph.h"
#include "wtime.h"
#include <queue>
#include <set>
#include <iterator>
#include <stdio.h>
#include <stdlib.h>
#include "herror.h"
#include <math.h>
#include "TC.cuh"
#include <assert.h>

int BUCKET_SIZE=1000;
using namespace std;

__device__ 
void d_display(int *a, int column,int row,int start)
{
		printf("\n");
		for(int i=0; i<row; i++)
		{
			for(int j=0;j<column;j++)
			{
				printf("%d\t",a[i*column+j+start]);
			}
			printf("\n");

		}		  
}

__device__
void kogge_sum(int *A,int len, int start)
{
    /* We require enough threads for this method */
    int step=log2f(len);
    //printf("Len: %d, Steps: %d, start: %d\n",len,step,start);
    int pos,i;
    for(i=0;i<step;i++)
    {
        pos=powf(2,i);
        int j=pos+threadIdx.x+start;
        while(j<(len+start))
        {
            int temp=A[j-pos];
            __syncthreads;
            A[j]+=temp;
            //printf("Write:%d , Read:%d , Written: %d\n",j,j-pos,A[j]);
            j+=blockDim.x;
        }
        //if(threadIdx.x==0){printf("\n\n");}
        __syncthreads();
	}
}

__device__
int linear_search(int neighbor,int *partition1, int *bin_count, int bin, int BIN_OFFSET, int BIN_START,int BUCKETS)
{
	int len = bin_count[bin+BIN_OFFSET];
	//printf("\nPartStart: %d\n",BIN_START);
	int i = bin + BIN_START;
	int step=0;
	while(step<len)
	{
		int test=partition1[i];
		//printf("Neighbor: %d, Test: %d\n",neighbor,test);
		if(test==neighbor)
		{
			return 1;
		}
		else
		{
			i+=BUCKETS;
		}
		step+=1;
	}
	return 0;
}

__device__
int merge(int *A, int *B, int ai, int bi, int l1_e, int l2_e,int steps)
{
	/*Reminder: As the partition is coalesced, accessing next element in each partition would require: next element --> prev + Warpsize */
	int WARPSIZE=64;
	int count = 0;
	int steps_count=0;
	while((ai<=l1_e) && (bi<=l2_e))
	{
		steps_count+=1;
		//printf("\nAI: %d, value: %d \t",ai, A[ai]);
		//printf("BI: %d, value: %d \t",bi, B[bi]);
		if(A[ai]>B[bi])
		{
			bi+=WARPSIZE;	
		}
		else if(A[ai]<B[bi])
		{
			ai+=WARPSIZE;
		}
		else
		{
			count+=1;
			ai+=WARPSIZE;
			bi+=WARPSIZE;
		}
		//printf("\n");
		__syncthreads();
	}
	//printf("Thread: %d, count: %d \n",threadIdx.x,count);
	return count;
}

int binary_search(long long start,long long end,int value, long long *arr)
{
    //printf("low:%d,high:%d,value:%f\n",start,end,value); 
    long long low=start;
    long long high=end;
    long long index=start;
    while (low<=high)
    {
	index=((low+high)/2);
        if (value<arr[index])
		{
            //set high to index-1
            high= index-1;
	    //printf("high:%d\n",high);
        }
        else if (value>arr[index])
        {
            // set low to index+1
            low = index+1;
            //printf("low:%d\n",low);

	}
        else
        {
            break;
        } 
	}
	//printf("Vaue: %d,Found: %d\n",value,arr[index]);
    return index;
}

__device__
int max_count(int *bin_count,int start,int end,int len)
{
	int max_count =bin_count[start];
	int min_count=bin_count[start];
	int zero_count=0;
	for (int i=start;i<end;i++)
	{
		if(bin_count[i]>max_count)
		{
			max_count=bin_count[i];			
		}
		if(bin_count[i]<min_count)
		{
			min_count=bin_count[i];
		}
		if(bin_count[i]==0)
		{
			zero_count+=1;
		}
	}
	// printf("%d,%d,%d\n",zero_count,max_count,len);
	return max_count;
}

void graph_reordering(graph *graph_temp)
{

}

__global__ void
warp_hash_count(vertex_t* adj_list, index_t* beg_pos, vertex_t* edge_list, int edge_count, int vertex_count,int edge_list_count, int *partition,unsigned long long *GLOBAL_COUNT,long long E_START, long long E_END, int device, int BUCKETS, int G_BUCKET_SIZE, int T_Group)
{
	// Uncomment the lines below and change partition to Gpartition for using shared version
	int *part;
	int S_BUCKET_SIZE=320;
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	int WARPSIZE=T_Group;
	int __shared__ bin_count[256*4];
	//int __shared__ partition[160*4];
	int PER_BLOCK_WARP=blockDim.x/WARPSIZE;
	int G_WARPID= tid/WARPSIZE;
	int WARPID = threadIdx.x/WARPSIZE;
	int __shared__ G_counter;
	G_counter=0;
	int P_counter=0;
	int BINsize = BUCKETS*G_BUCKET_SIZE;
	//int BINsize = BUCKETS*5;
	int BIN_START = G_WARPID*BINsize;
	//int BIN_START = WARPID*BINsize;
	long long i=G_WARPID*2;
	long long RANGE= E_END-E_START;
	int BIN_OFFSET= WARPID*BUCKETS;
	//for(int i=0;i<edge_list_count; i+=2)
	//TODO: Static assignment to dynamic assignment of edges 
	// unsigned long long TT=0,HT=0,IT=0;
	// unsigned long long __shared__ G_TT,G_HT,G_IT;
	// G_TT=0,G_HT=0,G_IT=0;
	while(i<( RANGE))
	{
		//if(threadIdx.x%32==0){printf("Warp:%d, G_WArp: %d,i: %d \n",WARPID,G_WARPID,i);}
		//if (device==1){printf("Device: %d, i: %d\n",device,i);}
		/* TODO: Divide edge list to multiple blocks*/
		// unsigned long long start_time=clock64();
		int destination = edge_list[i];
		int source = edge_list[i+1];
		int N1_start=beg_pos[destination];
		int N1_end= beg_pos[destination+1];	
		int L1= N1_end-N1_start;
		int N2_start=beg_pos[source];
		int N2_end= beg_pos[source+1];	
		int L2= N2_end-N2_start;
		
		// if ((L1==0))
		// {
		// 	//printf("continue %d\n",i);
		// 	continue;
		// }
		// // N2 is for hashing and N1 is lookup
		if(L1>L2)
		{
			int temp= N1_start;
			N1_start= N2_start;
			N2_start=temp;
			temp=N1_end;
			N1_end=N2_end;
			N2_end=temp;
			temp=L2;
			L2=L1;
			L1=temp;
		}
		
		// unsigned long long hash_start=clock64();
		int id=threadIdx.x%WARPSIZE+BIN_OFFSET;
		int end = BIN_OFFSET+BUCKETS;
		//if(threadIdx.x%32==0){printf("End: %d\n",end);}
		// We can remove this line
		
		__syncwarp();
		while(id<(end))
		{
			bin_count[id]=0;
			//printf("BIN: %d\n",id);
			id+=WARPSIZE;
		}
		int start=threadIdx.x%WARPSIZE + N2_start;
		// BIN_OFFSET is for count of number of element of each bin for all 4 warps
		
		__syncwarp();
		// Hash one list 
		while(start<N2_end)
		{
			int temp= adj_list[start];
			int bin=temp%BUCKETS;
			int index=atomicAdd(&bin_count[bin+BIN_OFFSET],1);
			partition[index*BUCKETS+ bin + BIN_START]=temp;
			//{printf("thread: %d,warp:%d, write: %d bin %d, index %d  at: %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}	
			start+=WARPSIZE;
		}
		__syncwarp();
		// unsigned long long hash_time=clock64()-hash_start;
		//int max_len_collision= max_count(bin_count,BIN_OFFSET,BIN_OFFSET+BUCKETS,L2);
		
		// unsigned long long intersection_start=clock64();
		start=threadIdx.x%WARPSIZE + N1_start;
		int count;
		//if(threadIdx.x==32){printf("start: %d, BIN_OFFSET: %d\n",start,BIN_OFFSET);}
		//P_counter=0;
		while(start<N1_end)
		{
			count=0;
			int neighbor=adj_list[start];
			int bin=neighbor%BUCKETS;
			count=linear_search(neighbor,partition,bin_count,bin,BIN_OFFSET,BIN_START,BUCKETS);
			P_counter+=count;
			start+=WARPSIZE;
			//printf("Tid: %d, Search:%d\n",threadIdx.x,neighbor);
		}
		//atomicAdd(&GLOBAL_COUNT[0],P_counter);
		
		__syncwarp();
		// unsigned long long intersection_time=clock64()-intersection_start;
		// if(threadIdx.x%32==0){printf("I: %d, Start:%d, End:%d, Count:%d\n",i,vertex,vertex1,G_counter);}
		i+=gridDim.x*PER_BLOCK_WARP*2;
		// unsigned long long total_time=clock64()-start_time;
		// if(threadIdx.x%32==0){
		// 	// printf("%d %d %d\n",total_time, hash_time, intersection_time);
		// 	TT+=total_time;
		// 	HT+=hash_time;
		// 	IT+=intersection_time;
		// }
	}
	atomicAdd(&G_counter,P_counter);
	// atomicAdd(&G_HT,HT);
	// atomicAdd(&G_TT,TT);
	// atomicAdd(&G_IT,IT);
	__syncthreads();
	if(threadIdx.x==0)
	{
		// printf("%d\n",G_TT);
		atomicAdd(&GLOBAL_COUNT[0],G_counter);
		// atomicAdd(&GLOBAL_COUNT[1],G_TT);
		// atomicAdd(&GLOBAL_COUNT[2],G_HT);
		// atomicAdd(&GLOBAL_COUNT[3],G_IT);
	}
	
	//if(threadIdx.x==0){printf("Device: %d, Count:%d\n",device,GLOBAL_COUNT[0]);}

}

__global__ void
CTA_hash_count(vertex_t* adj_list, index_t* beg_pos, vertex_t* edge_list, int edge_count, int vertex_count,int edge_list_count, int *partition,unsigned long long *GLOBAL_COUNT,int E_START, int E_END, int device, int BUCKETS, int BUCKET_SIZE,int T_Group)
{
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	int WARPSIZE=128;
	int __shared__ bin_count[512];
	int G_WARPID= blockIdx.x;
	int WARPID = blockIdx.x;
	int __shared__ G_counter;
	G_counter=0;
	int P_counter=0;
	int BINsize = BUCKETS*BUCKET_SIZE;
	int i=G_WARPID*2;
	int RANGE= E_END-E_START;
	int BIN_START = G_WARPID*BINsize;
	int divid=vertex_count/BUCKETS;
	int max_len_collision;
	//for(int i=0;i<edge_list_count; i+=2)
	//TODO: Static assignment to dynamic assignment of edges 
	
	while(i<( RANGE))
	{

		/* TODO: Divide edge list to multiple blocks*/
		int destination = edge_list[i];
		int source = edge_list[i+1];
		int N1_start=beg_pos[destination];
		int N1_end= beg_pos[destination+1];	
		int L1= N1_end-N1_start;
		int N2_start=beg_pos[source];
		int N2_end= beg_pos[source+1];	
		int L2= N2_end-N2_start;

		// N2 is for hashing and N1 is lookup
		if(L1>L2)
		{
			int temp= N1_start;
			N1_start= N2_start;
			N2_start=temp;
			temp=N1_end;
			N1_end=N2_end;
			N2_end=temp;
			temp=L2;
			L2=L1;
			L1=temp;
		}
		

		int id=threadIdx.x;
		int end = BUCKETS;

		while(id<(end))
		{
			bin_count[id]=0;
			id+=blockDim.x;
		}
		__syncthreads();
		int start=threadIdx.x + N2_start;
		
		// Hash one list 
		while(start<N2_end)
		{
			int temp= adj_list[start];
			int bin=temp%BUCKETS;
			int index=atomicAdd(&bin_count[bin],1);
			partition[index*BUCKETS+ bin + BIN_START]=temp;
			//{printf("thread: %d,warp:%d, write: %d bin %d, index %d  at: %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}	
			start+=blockDim.x;
		}
		__syncthreads();
		if (threadIdx.x==0)
		{
			max_len_collision= max_count(bin_count,0,0+BUCKETS,L2);
			// printf("max_len_collision: %d\n",max_len_collision );
		}
		__syncthreads();
		start=threadIdx.x + N1_start;
		int count;
		//if(threadIdx.x==32){printf("start: %d, BIN_OFFSET: %d\n",start,BIN_OFFSET);}
		//P_counter=0;
		while(start<N1_end)
		{
			count=0;
			int neighbor=adj_list[start];
			int bin=neighbor%BUCKETS;
			count=linear_search(neighbor,partition,bin_count,bin,0,BIN_START,BUCKETS);
			P_counter+=count;
			start+=blockDim.x;
			//printf("Tid: %d, Search:%d\n",threadIdx.x,neighbor);
		}
		//atomicAdd(&GLOBAL_COUNT[0],P_counter);
		
		//if(threadIdx.x%32==0){printf("I: %d, Start:%d, End:%d, Count:%d\n",i,vertex,vertex1,G_counter);}
		i+=gridDim.x*2;
		
	}
	atomicAdd(&G_counter,P_counter);
	__syncthreads();
	if(threadIdx.x==0){atomicAdd(&GLOBAL_COUNT[0],max_len_collision);}
	
	//if(threadIdx.x==0){printf("Device: %d, Count:%d\n",device,GLOBAL_COUNT[0]);}
}

struct arguments Triangle_count(int rank, char name[100], struct arguments args, int total_process,int n_threads , int n_blocks, int BUCKETS, int select_thread_group, int select_partition)
{

	// printf("---------------Here----------------");
	int T_Group= 32;
	int PER_BLOCK_WARP= n_threads/T_Group;
	int total=n_blocks*PER_BLOCK_WARP*BUCKETS*BUCKET_SIZE;
    unsigned long long *counter=(unsigned long long *)malloc(sizeof(unsigned long long)*10);
	string json_file 	= name;
	 graph *graph_d 
		 = new graph	(json_file);
	 graph *graph_b 
	     = new graph	(json_file);

	//printf("Graph Adj Read: %d",graph_d->adj_list[10]); 
	//int N_GPUS=argv[1];
	int deviceCount;
	HRR(hipGetDeviceCount(&deviceCount));
	// cout<<deviceCount<<endl;
	//fprintf(stderr,"----------------Device count: %d\n",deviceCount);
	//hipSetDevice();
	HRR(hipSetDevice(rank%deviceCount));
	//hipDeviceProp_t devProp;
	//HRR(hipGetDeviceProperties(&devProp, rank));
	index_t vertex_count=	graph_d-> vert_count;
	index_t edge_count= graph_d-> edge_count;
	index_t edge_list_count= graph_d-> edge_list_count;
	index_t edges= edge_list_count>>1;
	/* Preprocessing Step to calculate the ratio */
	long long *prefix=(long long *)malloc(sizeof(long long)*edges);
	long long temp;
	for(int i=0;i<edge_list_count;i+=2)
	 {
	 	int destination = graph_d->edge_list[i];
	 	int source = graph_d->edge_list[i+1];
	 	int N1_start=graph_d->beg_pos[destination];
	 	int N1_end= graph_d->beg_pos[destination+1];
	 	int L1= N1_end-N1_start;
	 	int N2_start= graph_d->beg_pos[source];
	 	int N2_end= graph_d->beg_pos[source+1];	
	 	int L2= N2_end-N2_start;
	 	int sum=L1+L2;
		if(i==0)
		{
			temp=0;
		}
		else
		{
			temp =  sum +prefix[(i>>1)-1];
		}
		prefix[i>>1]= temp;
	 	//printf("vertexA: %d, D1: %d, vertexB: %d, D2: %d, Degree: %d, prefix: %d\n",vertex,L1,vertex1,L2,L1+L2,temp);
	 }
	//  cout<<"edge_list_count OK"<<endl;
	int total_degree= temp;
	 //printf("total degree: %d,total edges: %d, E_END: %d,E_start:%d, size: %d, rank: %d\n",temp,edges,E_END,E_START,SIZE,rank);
	int SIZE,ratio;
	long long E_END,E_START;
	//-------------------------------------------//
	if(select_partition==1)
	{
		SIZE = (total_degree/total_process);
		E_END= binary_search(0,edge_count,SIZE*(rank+1),prefix);
		E_START= binary_search(0,edge_count,SIZE*rank,prefix);
		E_END=E_END<<1;
		E_START=E_START<<1;
	}
	//--------------------------------------------//
	
	else
	{
		ratio=2*(edges/total_process);
		E_START=rank*ratio;
		E_END=E_START+ratio;
		SIZE= prefix[E_END/2]-prefix[E_START/2];
	}
	//--------------------------------------------//
	// E_START=0;E_END=edge_list_count;
	// cout<<"partition OK!"<<endl;
	// cout<<E_START<<' '<<E_END<<endl;
	assert(E_END>E_START);
	//fprintf(stderr,"Rank: %d, Devicecount: %d,  Start: %d, End: %d, Selected: %d\n",rank,deviceCount,E_START,E_END,(rank%deviceCount));
	if(rank==(total_process-1)){E_END=edge_list_count;}
	int *hash,* BIN_MEM;
	unsigned long long *GLOBAL_COUNT;
	index_t *d_beg_pos;
	vertex_t *d_adj_list,*d_edge_list;
	float memory_req = (sizeof(int)*total + sizeof(index_t)*(vertex_count+1)+ sizeof(vertex_t)*(edge_count)+sizeof(vertex_t)*(E_END-E_START+1))/(1024*1024);
	HRR(hipMalloc((void **) &GLOBAL_COUNT,sizeof(unsigned long long)*10));
	HRR(hipMalloc((void **) &BIN_MEM,sizeof(int)*total));
	HRR(hipMalloc((void **) &d_beg_pos,sizeof(index_t)*(vertex_count+1)));
	HRR(hipMalloc((void **) &d_adj_list,sizeof(vertex_t)*(edge_count)));
	HRR(hipMalloc((void **) &d_edge_list,sizeof(vertex_t)*(E_END-E_START+1))); // Swap edge list count with Eend - estart; --> gives error; may add some more
	
	
	HRR(hipMemcpy(d_edge_list,graph_d->edge_list+E_START,sizeof(vertex_t)*(E_END-E_START+1), hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_beg_pos,graph_d->beg_pos,sizeof(index_t)*(vertex_count+1), hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_adj_list,graph_d->adj_list,sizeof(vertex_t)*edge_count, hipMemcpyHostToDevice));

	double t1=wtime();
	double cmp_time;
	if(select_thread_group==1)
	{
		double time_start=wtime();
		CTA_hash_count<<<n_blocks,n_threads>>>(d_adj_list, d_beg_pos, d_edge_list, edge_count, vertex_count,edge_list_count, BIN_MEM,GLOBAL_COUNT,E_START,E_END,rank,BUCKETS,BUCKET_SIZE, T_Group);
		HRR(hipDeviceSynchronize());
    	cmp_time = wtime()-time_start;	
	}
	else
	{
		double time_start=wtime();
		warp_hash_count<<<n_blocks,n_threads>>>(d_adj_list, d_beg_pos, d_edge_list, edge_count, vertex_count,edge_list_count, BIN_MEM,GLOBAL_COUNT,E_START,E_END,rank,BUCKETS,BUCKET_SIZE, T_Group);
		HRR(hipDeviceSynchronize());
    	cmp_time = wtime()-time_start;
	}
	HRR(hipMemcpy(counter,GLOBAL_COUNT,sizeof(unsigned long long)*10, hipMemcpyDeviceToHost));
	//printf("Edges: %d,Start: %d, End: %d, Rank: %d,ratio:%d, Triangle: %d, Time: %f\n",E_END-E_START,E_START,E_END,rank,SIZE,counter[0],cmp_time);
	args.time=cmp_time;
	args.count=counter[0];
	args.edge_count=edges;
	args.degree= SIZE;
	args.vertices= vertex_count-1;
	return args;
}    

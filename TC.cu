#include "hip/hip_runtime.h"
#include <iostream>
#include "graph.h"
#include "wtime.h"
#include <queue>
#include <set>
#include <iterator>
#include <stdio.h>
#include <stdlib.h>
#include "herror.h"
#include <math.h>
#include "TC.cuh"
#include <assert.h>

int BUCKET_SIZE=100;
using namespace std;

__device__ 
void d_display(int *a, int column,int row,int start)
{
		printf("\n");
		for(int i=0; i<row; i++)
		{
			for(int j=0;j<column;j++)
			{
				printf("%d\t",a[i*column+j+start]);
			}
			printf("\n");

		}		  
}

__device__
void kogge_sum(int *A,int len, int start)
{
    /* We require enough threads for this method */
    int step=log2f(len);
    //printf("Len: %d, Steps: %d, start: %d\n",len,step,start);
    int pos,i;
    for(i=0;i<step;i++)
    {
        pos=powf(2,i);
        int j=pos+threadIdx.x+start;
        while(j<(len+start))
        {
            int temp=A[j-pos];
            __syncthreads;
            A[j]+=temp;
            //printf("Write:%d , Read:%d , Written: %d\n",j,j-pos,A[j]);
            j+=blockDim.x;
        }
        //if(threadIdx.x==0){printf("\n\n");}
        __syncthreads();
	}
}

__device__
int linear_search(int neighbor,int *partition1, int *bin_count, int bin, int BIN_OFFSET, int BIN_START,int BUCKETS)
{
	int len = bin_count[bin+BIN_OFFSET];
	//printf("\nPartStart: %d\n",BIN_START);
	int i = bin + BIN_START;
	int step=0;
	while(step<len)
	{
		int test=partition1[i];
		//printf("Neighbor: %d, Test: %d\n",neighbor,test);
		if(test==neighbor)
		{
			return 1;
		}
		else
		{
			i+=BUCKETS;
		}
		step+=1;
	}
	return 0;
}

__device__
int merge(int *A, int *B, int ai, int bi, int l1_e, int l2_e,int steps)
{
	/*Reminder: As the partition is coalesced, accessing next element in each partition would require: next element --> prev + Warpsize */
	int WARPSIZE=64;
	int count = 0;
	int steps_count=0;
	while((ai<=l1_e) && (bi<=l2_e))
	{
		steps_count+=1;
		//printf("\nAI: %d, value: %d \t",ai, A[ai]);
		//printf("BI: %d, value: %d \t",bi, B[bi]);
		if(A[ai]>B[bi])
		{
			bi+=WARPSIZE;	
		}
		else if(A[ai]<B[bi])
		{
			ai+=WARPSIZE;
		}
		else
		{
			count+=1;
			ai+=WARPSIZE;
			bi+=WARPSIZE;
		}
		//printf("\n");
		__syncthreads();
	}
	//printf("Thread: %d, count: %d \n",threadIdx.x,count);
	return count;
}

int binary_search(int start,int end,int value, int *arr)
{
    //printf("low:%d,high:%d,value:%f\n",start,end,value); 
    int low=start;
    int high=end;
    int index=start;
    while (low<=high)
    {
	index=((low+high)/2);
        if (value<arr[index])
		{
            //set high to index-1
            high= index-1;
	    //printf("high:%d\n",high);
        }
        else if (value>arr[index])
        {
            // set low to index+1
            low = index+1;
            //printf("low:%d\n",low);

	}
        else
        {
            break;
        } 
	}
	//printf("Vaue: %d,Found: %d\n",value,arr[index]);
    return index;
}

__device__
int max_count(int *bin_count,int start,int end,int len)
{
	int max_count =bin_count[start];
	int min_count=bin_count[start];
	int zero_count=0;
	for (int i=start;i<end;i++)
	{
		if(bin_count[i]>max_count)
		{
			max_count=bin_count[i];			
		}
		if(bin_count[i]<min_count)
		{
			min_count=bin_count[i];
		}
		if(bin_count[i]==0)
		{
			zero_count+=1;
		}
	}
	printf("%d,%d,%d\n",zero_count,max_count,len);
	return max_count;
}

void graph_reordering(graph *graph_temp)
{

}

__global__ void
warp_hash_count(vertex_t* adj_list, index_t* beg_pos, vertex_t* edge_list, int edge_count, int vertex_count,int edge_list_count, int *partition,int *GLOBAL_COUNT,int E_START, int E_END, int device, int BUCKETS, int G_BUCKET_SIZE, int T_Group)
{
	// Uncomment the lines below and change partition to Gpartition for using shared version
	int *part;
	int S_BUCKET_SIZE=320;
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	int WARPSIZE=T_Group;
	int __shared__ bin_count[256*4];
	//int __shared__ partition[160*4];
	int PER_BLOCK_WARP=blockDim.x/WARPSIZE;
	int G_WARPID= tid/WARPSIZE;
	int WARPID = threadIdx.x/WARPSIZE;
	int __shared__ G_counter;
	G_counter=0;
	int P_counter=0;
	int BINsize = BUCKETS*G_BUCKET_SIZE;
	//int BINsize = BUCKETS*5;
	int BIN_START = G_WARPID*BINsize;
	//int BIN_START = WARPID*BINsize;
	int i=G_WARPID*2;
	int RANGE= E_END-E_START;
	int BIN_OFFSET= WARPID*BUCKETS;
	//for(int i=0;i<edge_list_count; i+=2)
	//TODO: Static assignment to dynamic assignment of edges 
	
	while(i<( RANGE))
	{
		//if(threadIdx.x%32==0){printf("Warp:%d, G_WArp: %d,i: %d \n",WARPID,G_WARPID,i);}
		//if (device==1){printf("Device: %d, i: %d\n",device,i);}
		/* TODO: Divide edge list to multiple blocks*/
		int destination = edge_list[i];
		int source = edge_list[i+1];
		int N1_start=beg_pos[destination];
		int N1_end= beg_pos[destination+1];	
		int L1= N1_end-N1_start;
		int N2_start=beg_pos[source];
		int N2_end= beg_pos[source+1];	
		int L2= N2_end-N2_start;
		
		// if ((L1==0))
		// {
		// 	//printf("continue %d\n",i);
		// 	continue;
		// }
		// // N2 is for hashing and N1 is lookup
		if(L1>L2)
		{
			int temp= N1_start;
			N1_start= N2_start;
			N2_start=temp;
			temp=N1_end;
			N1_end=N2_end;
			N2_end=temp;
			temp=L2;
			L2=L1;
			L1=temp;
		}
		// if(L2<320)
		// {
		// 	part=partition
		// }
		//if(threadIdx.x==0 && device==1) {printf("L_N1: %d, L_N2:%d \n",L1,L2);}
		//printf("Length of N2: %d\n",L2);
		int id=threadIdx.x%WARPSIZE+BIN_OFFSET;
		int end = BIN_OFFSET+BUCKETS;
		//if(threadIdx.x%32==0){printf("End: %d\n",end);}
		// We can remove this line
		while(id<(end))
		{
			bin_count[id]=0;
			//printf("BIN: %d\n",id);
			id+=WARPSIZE;
		}
		int start=threadIdx.x%WARPSIZE + N2_start;
		// BIN_OFFSET is for count of number of element of each bin for all 4 warps
		
		// Hash one list 
		while(start<N2_end)
		{
			int temp= adj_list[start];
			int bin=temp%BUCKETS;
			int index=atomicAdd(&bin_count[bin+BIN_OFFSET],1);
			partition[index*BUCKETS+ bin + BIN_START]=temp;
			//{printf("thread: %d,warp:%d, write: %d bin %d, index %d  at: %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}	
			start+=WARPSIZE;
		}
		//int max_len_collision= max_count(bin_count,BIN_OFFSET,BIN_OFFSET+BUCKETS,L2);
		
		start=threadIdx.x%WARPSIZE + N1_start;
		int count;
		//if(threadIdx.x==32){printf("start: %d, BIN_OFFSET: %d\n",start,BIN_OFFSET);}
		//P_counter=0;
		while(start<N1_end)
		{
			count=0;
			int neighbor=adj_list[start];
			int bin=neighbor%BUCKETS;
			count=linear_search(neighbor,partition,bin_count,bin,BIN_OFFSET,BIN_START,BUCKETS);
			P_counter+=count;
			start+=WARPSIZE;
			//printf("Tid: %d, Search:%d\n",threadIdx.x,neighbor);
		}
		//atomicAdd(&GLOBAL_COUNT[0],P_counter);
		
		//if(threadIdx.x%32==0){printf("I: %d, Start:%d, End:%d, Count:%d\n",i,vertex,vertex1,G_counter);}
		i+=gridDim.x*PER_BLOCK_WARP*2;
		
	}
	atomicAdd(&G_counter,P_counter);
	__syncthreads();
	if(threadIdx.x==0){atomicAdd(&GLOBAL_COUNT[0],G_counter);}
	
	//if(threadIdx.x==0){printf("Device: %d, Count:%d\n",device,GLOBAL_COUNT[0]);}
}

__global__ void
CTA_hash_count(vertex_t* adj_list, index_t* beg_pos, vertex_t* edge_list, int edge_count, int vertex_count,int edge_list_count, int *partition,int *GLOBAL_COUNT,int E_START, int E_END, int device, int BUCKETS, int BUCKET_SIZE,int T_Group)
{
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	int WARPSIZE=128;
	int __shared__ bin_count[512];
	int G_WARPID= blockIdx.x;
	int WARPID = blockIdx.x;
	int __shared__ G_counter;
	G_counter=0;
	int P_counter=0;
	int BINsize = BUCKETS*BUCKET_SIZE;
	int i=G_WARPID*2;
	int RANGE= E_END-E_START;
	int BIN_START = G_WARPID*BINsize;
	//for(int i=0;i<edge_list_count; i+=2)
	//TODO: Static assignment to dynamic assignment of edges 
	
	while(i<( RANGE))
	{
		//if(threadIdx.x%32==0){printf("Warp:%d, G_WArp: %d,i: %d \n",WARPID,G_WARPID,i);}
		//if (device==1){printf("Device: %d, i: %d\n",device,i);}
		/* TODO: Divide edge list to multiple blocks*/
		int destination = edge_list[i];
		int source = edge_list[i+1];
		int N1_start=beg_pos[destination];
		int N1_end= beg_pos[destination+1];	
		int L1= N1_end-N1_start;
		int N2_start=beg_pos[source];
		int N2_end= beg_pos[source+1];	
		int L2= N2_end-N2_start;

		// N2 is for hashing and N1 is lookup
		if(L1>L2)
		{
			int temp= N1_start;
			N1_start= N2_start;
			N2_start=temp;
			temp=N1_end;
			N1_end=N2_end;
			N2_end=temp;
			temp=L2;
			L2=L1;
			L1=temp;
		}
		
		//if(threadIdx.x==0 && device==1) {printf("L_N1: %d, L_N2:%d \n",L1,L2);}
		//printf("Length of N2: %d\n",L2);
		int id=threadIdx.x;
		int end = BUCKETS;
		//if(threadIdx.x%32==0){printf("End: %d\n",end);}
		// We can remove this line
		while(id<(end))
		{
			bin_count[id]=0;
			//printf("BIN: %d\n",id);
			id+=blockDim.x;
		}
		__syncthreads();
		int start=threadIdx.x + N2_start;
		// BIN_OFFSET is for count of number of element of each bin for all 4 warps
		
		// Hash one list 
		while(start<N2_end)
		{
			int temp= adj_list[start];
			int bin=temp%BUCKETS;
			int index=atomicAdd(&bin_count[bin],1);
			partition[index*BUCKETS+ bin + BIN_START]=temp;
			//{printf("thread: %d,warp:%d, write: %d bin %d, index %d  at: %d\n",threadIdx.x,WARPID,temp,bin,index,(index*WARPSIZE+bin+BIN_START));}	
			start+=blockDim.x;
		}
		//int max_len_collision= max_count(bin_count,BIN_OFFSET,BIN_OFFSET+BUCKETS,L2);
		__syncthreads();
		start=threadIdx.x + N1_start;
		int count;
		//if(threadIdx.x==32){printf("start: %d, BIN_OFFSET: %d\n",start,BIN_OFFSET);}
		//P_counter=0;
		while(start<N1_end)
		{
			count=0;
			int neighbor=adj_list[start];
			int bin=neighbor%BUCKETS;
			count=linear_search(neighbor,partition,bin_count,bin,0,BIN_START,BUCKETS);
			P_counter+=count;
			start+=blockDim.x;
			//printf("Tid: %d, Search:%d\n",threadIdx.x,neighbor);
		}
		i+=gridDim.x*2;
		
	}
	atomicAdd(&G_counter,P_counter);
	__syncthreads();
	if(threadIdx.x==0){atomicAdd(&GLOBAL_COUNT[0],G_counter);}
	
	//if(threadIdx.x==0){printf("Device: %d, Count:%d\n",device,GLOBAL_COUNT[0]);}
}

struct arguments Triangle_count(int rank, char name[100], struct arguments args, int total_process,int n_threads , int n_blocks, int BUCKETS, int select_thread_group, int select_partition)
{

	//fprintf(stderr,"---------------Here----------------");
	int T_Group= 32;
	int PER_BLOCK_WARP= n_threads/T_Group;
	int total=n_blocks*PER_BLOCK_WARP*BUCKETS*BUCKET_SIZE;
    int *counter=(int *)malloc(sizeof(int));
	string json_file 	= name;
	 graph *graph_d 
		 = new graph	(json_file);
	 graph *graph_b 
	     = new graph	(json_file);

	//printf("Graph Adj Read: %d",graph_d->adj_list[10]); 
	//int N_GPUS=argv[1];
	int deviceCount;
	HRR(hipGetDeviceCount(&deviceCount));
	//fprintf(stderr,"----------------Device count: %d\n",deviceCount);
	  
	//hipSetDevice();
	HRR(hipSetDevice(rank%deviceCount));
	//hipDeviceProp_t devProp;
	//HRR(hipGetDeviceProperties(&devProp, rank));
	index_t vertex_count=	graph_d-> vert_count;
	index_t edge_count= graph_d-> edge_count;
	index_t edge_list_count= graph_d-> edge_list_count;
	index_t edges= edge_list_count>>1;
	/* Preprocessing Step to calculate the ratio  for workload balancing based upon the combined degree of each edge.*/
	int *prefix=(int *)malloc(sizeof(int)*edges);
	int temp;
	for(int i=0;i<edge_list_count;i+=2)
	 {
	 	int destination = graph_d->edge_list[i];
	 	int source = graph_d->edge_list[i+1];
	 	int N1_start=graph_d->beg_pos[destination];
	 	int N1_end= graph_d->beg_pos[destination+1];
	 	int L1= N1_end-N1_start;
	 	int N2_start= graph_d->beg_pos[source];
	 	int N2_end= graph_d->beg_pos[source+1];	
	 	int L2= N2_end-N2_start;
	 	int sum=L1+L2;
		if(i==0)
		{
			temp=0;
		}
		else
		{
			temp =  sum +prefix[(i>>1)-1];
		}
		prefix[i>>1]= temp;
	 	//printf("vertexA: %d, D1: %d, vertexB: %d, D2: %d, Degree: %d, prefix: %d\n",vertex,L1,vertex1,L2,L1+L2,temp);
	 }
	int total_degree= temp;
	 //printf("total degree: %d,total edges: %d, E_END: %d,E_start:%d, size: %d, rank: %d\n",temp,edges,E_END,E_START,SIZE,rank);
	int SIZE,E_END,E_START,ratio;
	
	//-------------------------------------------//
	if(select_partition==1)
	{
		SIZE = (total_degree/total_process);
		E_END= binary_search(0,edges,SIZE*(rank+1),prefix);
		E_START= binary_search(0,edges,SIZE*rank,prefix);
		E_END=E_END<<1;
		E_START=E_START<<1;
	}
	//--------------------------------------------//
	
	else
	{
		ratio=2*(edges/total_process);
		E_START=rank*ratio;
		E_END=E_START+ratio;
		SIZE= prefix[E_END/2]-prefix[E_START/2];
	}
	//--------------------------------------------//
	
	assert(E_END>E_START);
	//fprintf(stderr,"Rank: %d, Devicecount: %d,  Start: %d, End: %d, Selected: %d\n",rank,deviceCount,E_START,E_END,(rank%deviceCount));
	if(rank==(total_process-1)){E_END=edge_list_count;}
	int *hash,* BIN_MEM;
	int *GLOBAL_COUNT;
	index_t *d_beg_pos;
	vertex_t *d_adj_list,*d_edge_list;
	float memory_req = (sizeof(int)*total + sizeof(index_t)*(vertex_count+1)+ sizeof(vertex_t)*(edge_count)+sizeof(vertex_t)*(E_END-E_START+1))/(1024*1024);
	//fprintf(stderr,"-------------------GPU: %d, Memory required: %f MB\n",rank,memory_req);
	HRR(hipMalloc((void **) &GLOBAL_COUNT,sizeof(int)*2));
	HRR(hipMalloc((void **) &BIN_MEM,sizeof(int)*total));
	HRR(hipMalloc((void **) &d_beg_pos,sizeof(index_t)*(vertex_count+1)));
	HRR(hipMalloc((void **) &d_adj_list,sizeof(vertex_t)*(edge_count)));
	HRR(hipMalloc((void **) &d_edge_list,sizeof(vertex_t)*(E_END-E_START+1))); // Swap edge list count with Eend - estart; --> gives error; may add some more
	//fprintf(stderr,">>>>>>>>>>>>>>>>>Malloc:adj_list[10]: %d\n",graph_d->adj_list[10]);
	
	
	HRR(hipMemcpy(d_edge_list,graph_d->edge_list+E_START,sizeof(vertex_t)*(E_END-E_START+1), hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_beg_pos,graph_d->beg_pos,sizeof(index_t)*(vertex_count+1), hipMemcpyHostToDevice));
	HRR(hipMemcpy(d_adj_list,graph_d->adj_list,sizeof(vertex_t)*edge_count, hipMemcpyHostToDevice));
	//fprintf(stderr,">>>>>>>>>>>>>>>>>>>Memcopy completed");	
	//HRR(hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024));
	double t1=wtime();
	double cmp_time;
	if(select_thread_group==1)
	{
		double time_start=wtime();
		CTA_hash_count<<<n_blocks,n_threads>>>(d_adj_list, d_beg_pos, d_edge_list, edge_count, vertex_count,edge_list_count, BIN_MEM,GLOBAL_COUNT,E_START,E_END,rank,BUCKETS,BUCKET_SIZE, T_Group);
		HRR(hipDeviceSynchronize());
    	cmp_time = wtime()-time_start;	
	}
	else
	{
		double time_start=wtime();
		warp_hash_count<<<n_blocks,n_threads>>>(d_adj_list, d_beg_pos, d_edge_list, edge_count, vertex_count,edge_list_count, BIN_MEM,GLOBAL_COUNT,E_START,E_END,rank,BUCKETS,BUCKET_SIZE, T_Group);
		HRR(hipDeviceSynchronize());
    	cmp_time = wtime()-time_start;
	}
	HRR(hipMemcpy(counter,GLOBAL_COUNT,sizeof(int), hipMemcpyDeviceToHost));
	//printf("Edges: %d,Start: %d, End: %d, Rank: %d,ratio:%d, Triangle: %d, Time: %f\n",E_END-E_START,E_START,E_END,rank,SIZE,counter[0],cmp_time);
	args.time=cmp_time;
	args.count=counter[0];
	args.edge_count=edges;
	args.degree= SIZE;
	args.vertices= vertex_count-1;
	return args;
}    
